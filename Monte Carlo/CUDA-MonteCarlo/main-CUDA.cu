#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <vector>
#include <cmath>
#include <chrono>

// CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;
using namespace chrono;

// Function to integrate: f(x) = (1 + r)^(-t)
__device__ double functionToIntegrate(double x) {
    double r = 0.08; // Annual discount rate 
    double t = x; // Time in years 

    return pow(1.0 + r, -t);
}

__global__ void monteCarloIntegrationKernel(double* results, double lowerBound, double upperBound, int numSamples, unsigned int seed, double* totalSum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize random number generator
    hiprandState_t state;
    hiprand_init(seed, idx, 0, &state);

    double partialSum = 0.0;
    // Perform Monte Carlo integration for the assigned samples
    for (int i = idx; i < numSamples; i += blockDim.x * gridDim.x) {
        double x = lowerBound + (upperBound - lowerBound) * hiprand_uniform(&state);
        partialSum += functionToIntegrate(x);
    }

    // Store the partial sum in the results array
    results[idx] = partialSum;

    // Synchronize all threads to ensure all partial sums are stored
    __syncthreads();

    // Perform reduction to calculate the total sum
    if (idx == 0) {
        double total = 0.0;
        for (int i = 0; i < blockDim.x * gridDim.x; ++i) {
            total += results[i];
        }
        *totalSum = total;
    }
}

__global__ void performSimulationsKernel(double* returns, double initialInvestment, double expectedReturn, double volatility,
                                         int numSimulations, int investmentPeriod, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize random number generator
    hiprandState_t state;
    hiprand_init(seed, idx, 0, &state);

    // Perform simulations for the assigned index
    for (int i = idx; i < numSimulations; i += blockDim.x * gridDim.x) {
        double cumulativeReturn = 1.0;

        for (int j = 0; j < investmentPeriod; ++j) {
            double randomValue = hiprand_normal(&state);
            double investmentReturn = expectedReturn + volatility * randomValue;
            cumulativeReturn *= (1.0 + investmentReturn);
        }

        returns[i] = cumulativeReturn - 1.0;
    }
}

void calculateRiskMetrics(const vector<double>& finalReturns, double& averageReturn, double& standardDeviation,
                          double& minReturn, double& maxReturn) {
    double totalReturns = 0.0;
    minReturn = finalReturns[0];
    maxReturn = finalReturns[0];

    for (const auto& returnVal : finalReturns) {
        totalReturns += returnVal;
        if (returnVal < minReturn) {
            minReturn = returnVal;
        }
        if (returnVal > maxReturn) {
            maxReturn = returnVal;
        }
    }

    averageReturn = totalReturns / finalReturns.size();

    double variance = 0.0;
    for (const auto& returnVal : finalReturns) {
        variance += (returnVal - averageReturn) * (returnVal - averageReturn);
    }

    variance /= (finalReturns.size() - 1);
    standardDeviation = sqrt(variance);
}

double calcAverage(double* array, int iterations)
{
    double sum = 0.0;
    for (int i = 0; i < iterations; i++) {
        sum += array[iterations];
    }
    
    return sum;
}

void displayResults(double initialInvestment, double expectedReturn, double volatility, int investmentPeriod,
                    int numSimulations, double averageReturn, double standardDeviation, double minReturn,
                    double maxReturn, double integrationResult, double elapsedTime) {
    cout << "Risk Assessment Results:" << endl;
    cout << "--------------------------------------" << endl;
    cout << "Initial Investment: $" << initialInvestment << endl;
    cout << "Expected Annual Return: " << (expectedReturn * 100.0) << "%" << endl;
    cout << "Volatility (Annual Standard Deviation): " << (volatility * 100.0) << "%" << endl;
    cout << "Investment Period: " << investmentPeriod << " years" << endl;
    cout << "Number of Simulations: " << numSimulations << endl;
    cout << "--------------------------------------" << endl;
    cout << "Average Return: " << (averageReturn * 100.0) << "%" << endl;
    cout << "Standard Deviation of Returns: " << (standardDeviation * 100.0) << "%" << endl;
    cout << "Minimum Return: " << (minReturn * 100.0) << "%" << endl;
    cout << "Maximum Return: " << (maxReturn * 100.0) << "%" << endl;
    cout << "Monte Carlo Integration Result: " << integrationResult << endl;
    cout << "Elapsed Time: " << elapsedTime << " seconds" << endl;
}

int main() {
    // Define your input parameters
    double initialInvestment = 10000.0;
    double expectedReturn = 0.08;
    double volatility = 0.15;
    int investmentPeriod = 5;
    int numSimulations = 1000000;
    double lowerBound = 0.0;
    double upperBound = 1;
    int numThreads = 8;

    // Allocate memory for results
    double* returns = new double[numSimulations];
    double* integral = new double[numSimulations];

    // Set up CUDA memory
    double* d_returns;
    double* d_integral;
    hipMalloc((void**)&d_returns, numSimulations * sizeof(double));
    hipMalloc((void**)&d_integral, numSimulations * sizeof(double));

    double* integralResult;
    hipMalloc((void**)&integralResult, sizeof(double));

    // Set up CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch CUDA kernels in parallel
    hipEventRecord(start);

    // Create an array of CUDA streams
    hipStream_t* streams = new hipStream_t[numThreads];

    unsigned int seed = static_cast<unsigned int>(time(nullptr));
    // Launch parallel simulations and integration kernels using separate streams
    for (int i = 0; i < numThreads; ++i) {
        // Create a new CUDA stream for the current thread
        hipStreamCreate(&streams[i]);

        // Perform simulations on the device (GPU) using the current stream
        performSimulationsKernel << <1, 256, 0, streams[i] >> > (d_returns + i * numSimulations / numThreads,
            initialInvestment, expectedReturn, volatility,
            numSimulations / numThreads, investmentPeriod, seed + i);

        // Launch Monte Carlo integration kernel on the device (GPU) using the current stream
        monteCarloIntegrationKernel << < 1, 256, 0, streams[i] >> > (d_integral + i * numSimulations / numThreads,
            lowerBound, upperBound, numSimulations / numThreads, seed + i, integralResult);
    }

    // Synchronize all CUDA streams before copying results back to the host
    for (int i = 0; i < numThreads; ++i) {
        hipStreamSynchronize(streams[i]);
    }
    
    // Copy results from device to host
    hipMemcpy(returns, d_returns, numSimulations * sizeof(double), hipMemcpyDeviceToHost);
    //hipMemcpy(integral, d_integral, numSimulations * sizeof(double), hipMemcpyDeviceToHost);
    double totalSum;
    hipMemcpy(&totalSum, integralResult, sizeof(double), hipMemcpyDeviceToHost);
    double avgIntegral = (totalSum / numSimulations) * (upperBound - lowerBound);

    // Calculate risk metrics on the host (CPU)
    double averageReturn, standardDeviation, minReturn, maxReturn;
    calculateRiskMetrics(vector<double>(returns, returns + numSimulations), averageReturn, standardDeviation,
        minReturn, maxReturn);

    // Record stop time and calculate elapsed time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    // CUDA Calculate Time in ms, therefore divide by 1000
    elapsedTime /= 1000;

    // Display results
    displayResults(initialInvestment, expectedReturn, volatility, investmentPeriod, numSimulations,
        averageReturn, standardDeviation, minReturn, maxReturn, avgIntegral,
        elapsedTime);

    // Clean up CUDA resources
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_returns);
    delete[] returns;
    delete[] streams;

    system("pause");

    return 0;
}